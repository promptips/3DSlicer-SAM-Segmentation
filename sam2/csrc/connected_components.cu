#include "hip/hip_runtime.h"

// Copyright (c) Meta Platforms, Inc. and affiliates.
// All rights reserved.

// This source code is licensed under the license found in the
// LICENSE file in the root directory of this source tree.

// adapted from https://github.com/zsef123/Connected_components_PyTorch
// with license found in the LICENSE_cctorch file in the root directory.
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <torch/script.h>
#include <vector>

// 2d
#define BLOCK_ROWS 16
#define BLOCK_COLS 16

namespace cc2d {

template <typename T>
__device__ __forceinline__ unsigned char hasBit(T bitmap, unsigned char pos) {
  return (bitmap >> pos) & 1;
}

__device__ int32_t find(const int32_t* s_buf, int32_t n) {
  while (s_buf[n] != n)
    n = s_buf[n];
  return n;
}

__device__ int32_t find_n_compress(int32_t* s_buf, int32_t n) {
  const int32_t id = n;
  while (s_buf[n] != n) {
    n = s_buf[n];
    s_buf[id] = n;
  }
  return n;
}

__device__ void union_(int32_t* s_buf, int32_t a, int32_t b) {
  bool done;
  do {
    a = find(s_buf, a);
    b = find(s_buf, b);

    if (a < b) {
      int32_t old = atomicMin(s_buf + b, a);
      done = (old == b);
      b = old;
    } else if (b < a) {
      int32_t old = atomicMin(s_buf + a, b);
      done = (old == a);
      a = old;
    } else
      done = true;

  } while (!done);
}

__global__ void
init_labeling(int32_t* label, const uint32_t W, const uint32_t H) {
  const uint32_t row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
  const uint32_t col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
  const uint32_t idx = row * W + col;

  if (row < H && col < W)
    label[idx] = idx;
}

__global__ void
merge(uint8_t* img, int32_t* label, const uint32_t W, const uint32_t H) {
  const uint32_t row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
  const uint32_t col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
  const uint32_t idx = row * W + col;

  if (row >= H || col >= W)
    return;

  uint32_t P = 0;

  if (img[idx])
    P |= 0x777;
  if (row + 1 < H && img[idx + W])
    P |= 0x777 << 4;
  if (col + 1 < W && img[idx + 1])
    P |= 0x777 << 1;

  if (col == 0)
    P &= 0xEEEE;
  if (col + 1 >= W)
    P &= 0x3333;
  else if (col + 2 >= W)
    P &= 0x7777;

  if (row == 0)
    P &= 0xFFF0;
  if (row + 1 >= H)
    P &= 0xFF;

  if (P > 0) {
    // If need check about top-left pixel(if flag the first bit) and hit the
    // top-left pixel
    if (hasBit(P, 0) && img[idx - W - 1]) {
      union_(label, idx, idx - 2 * W - 2); // top left block
    }

    if ((hasBit(P, 1) && img[idx - W]) || (hasBit(P, 2) && img[idx - W + 1]))
      union_(label, idx, idx - 2 * W); // top bottom block

    if (hasBit(P, 3) && img[idx + 2 - W])
      union_(label, idx, idx - 2 * W + 2); // top right block

    if ((hasBit(P, 4) && img[idx - 1]) || (hasBit(P, 8) && img[idx + W - 1]))
      union_(label, idx, idx - 2); // just left block
  }
}

__global__ void compression(int32_t* label, const int32_t W, const int32_t H) {
  const uint32_t row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
  const uint32_t col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
  const uint32_t idx = row * W + col;

  if (row < H && col < W)
    find_n_compress(label, idx);
}

__global__ void final_labeling(
    const uint8_t* img,
    int32_t* label,
    const int32_t W,
    const int32_t H) {
  const uint32_t row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
  const uint32_t col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
  const uint32_t idx = row * W + col;

  if (row >= H || col >= W)
    return;

  int32_t y = label[idx] + 1;

  if (img[idx])
    label[idx] = y;
  else
    label[idx] = 0;

  if (col + 1 < W) {
    if (img[idx + 1])
      label[idx + 1] = y;
    else