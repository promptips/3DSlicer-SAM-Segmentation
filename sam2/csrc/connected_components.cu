
// Copyright (c) Meta Platforms, Inc. and affiliates.
// All rights reserved.

// This source code is licensed under the license found in the
// LICENSE file in the root directory of this source tree.

// adapted from https://github.com/zsef123/Connected_components_PyTorch
// with license found in the LICENSE_cctorch file in the root directory.
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <torch/script.h>
#include <vector>

// 2d
#define BLOCK_ROWS 16
#define BLOCK_COLS 16

namespace cc2d {

template <typename T>
__device__ __forceinline__ unsigned char hasBit(T bitmap, unsigned char pos) {
  return (bitmap >> pos) & 1;
}

__device__ int32_t find(const int32_t* s_buf, int32_t n) {
  while (s_buf[n] != n)
    n = s_buf[n];
  return n;
}

__device__ int32_t find_n_compress(int32_t* s_buf, int32_t n) {
  const int32_t id = n;
  while (s_buf[n] != n) {
    n = s_buf[n];
    s_buf[id] = n;
  }
  return n;
}

__device__ void union_(int32_t* s_buf, int32_t a, int32_t b) {
  bool done;
  do {
    a = find(s_buf, a);
    b = find(s_buf, b);

    if (a < b) {